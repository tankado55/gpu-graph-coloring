#include "hip/hip_runtime.h"
﻿
#include ""
#include <iostream>
#include "coloring.h"
#include "utils/graph/graph_d.h"
#include "utils/common.h"
#include <hip/hip_cooperative_groups.h>

using namespace std;

#define THREADxBLOCK 128

Coloring* LubyGreedy(GraphStruct* graphStruct) {
	// set coloring struct

	Coloring* col;
	CHECK(hipMallocManaged(&col, sizeof(Coloring)));
	uint n = graphStruct->nodeCount;
	col->uncoloredNodes = true;

	// hipMalloc for arrays of struct Coloring
	CHECK(hipMallocManaged(&(col->coloring), n * sizeof(uint)));
	memset(col->coloring, 0, n);

	// allocate space on the GPU for the random states
	hiprandState_t* states;
	uint* weigths;
	hipMalloc((void**)&states, n * sizeof(hiprandState_t));
	hipMalloc((void**)&weigths, n * sizeof(uint));
	dim3 threads(THREADxBLOCK);
	dim3 blocks((graphStruct->nodeCount + threads.x - 1) / threads.x, 1, 1);
	uint seed = 0;
	init <<< blocks, threads >>> (seed, states, weigths, n);
	hipDeviceSynchronize();
	// start coloring (dyn. parall.)
	LubyJPcolorer(col, graphStruct, weigths);

	hipFree(states);
	hipFree(weigths);
	return col;
}

/**
 * find an IS
 */
__global__ void findIS(Coloring* col, GraphStruct* graphStruct, uint* weights) {
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= graphStruct->nodeCount)
		return;

	if (col->coloring[idx])
		return;

	uint offset = graphStruct->cumDegs[idx];
	uint deg = graphStruct->cumDegs[idx + 1] - graphStruct->cumDegs[idx];

	bool candidate = true;
	for (uint j = 0; j < deg; j++) {
		uint neighID = graphStruct->neighs[offset + j];
		if (!col->coloring[neighID] &&
			((weights[idx] < weights[neighID]) ||
				((weights[idx] == weights[neighID]) && idx < neighID))) {
			candidate = false;
		}
	}
	if (candidate) {
		col->coloring[idx] = col->numOfColors;
	}
	else
		col->uncoloredNodes = true;
}


/**
 *  this GPU kernel takes an array of states, and an array of ints, and puts a random int into each
 */
__global__ void init(uint seed, hiprandState_t* states, uint* numbers, uint n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > n)
		return;
	hiprand_init(seed, idx, 0, &states[idx]);
	numbers[idx] = hiprand(&states[idx]) % n * n;
}

__global__ void initLDF(GraphStruct* graphStruct, uint n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;

	uint degree = graphStruct->cumDegs[idx + 1] - graphStruct->cumDegs[idx];

	for (int i = 0; i < degree; ++i)
	{
		uint neighID = graphStruct->neighs[graphStruct->cumDegs[idx + i]];
		uint neighDegree = graphStruct->cumDegs[neighID + 1] - graphStruct->cumDegs[neighID]; // ottimizzabile su CPU
		if (degree > neighDegree)
		{
			atomicAdd(&graphStruct->inCount[neighID], 1);
		}
		else if (degree == neighDegree && idx > neighID)
		{
			atomicAdd(&graphStruct->inCount[neighID], 1);
		}
	}
}

void LDFColoring(GraphStruct* graphStruct)
{
	Coloring* coloring;
	CHECK(hipMallocManaged(&coloring, sizeof(Coloring)));
	uint n = graphStruct->nodeCount;
	coloring->uncoloredNodes = true;

	// hipMalloc for arrays of struct Coloring
	CHECK(hipMallocManaged(&(col->coloring), n * sizeof(uint)));
	memset(coloring->coloring, 0, n);

	dim3 threads(THREADxBLOCK);
	dim3 blocks((graphStruct->nodeCount + threads.x - 1) / threads.x, 1, 1);

	coloring->numOfColors = 0;
	while (coloring->uncoloredNodes) {
		coloring->uncoloredNodes = false;
		coloring->numOfColors++;
		findIS << < blocks, threads >> > (coloring, graphStruct);
		hipDeviceSynchronize();
	}
}


/**
 * Luby IS & Jones−Plassmann colorer
 */
void LubyJPcolorer(Coloring* col, GraphStruct* graphStruct, uint* weights) {
	dim3 threads(THREADxBLOCK);
	dim3 blocks((graphStruct->nodeCount + threads.x - 1) / threads.x, 1, 1);

	// loop on ISs covering the graph
	col->numOfColors = 0;
	while (col->uncoloredNodes) {
		col->uncoloredNodes = false;
		col->numOfColors++;
		findIS <<< blocks, threads >>> (col, graphStruct, weights);
		hipDeviceSynchronize();
	}
}


/**
 * Print the graph (verbose = 1 for "verbose print")
 * @param verbose print the complete graph
 */
void printColoring(Coloring* col, GraphStruct* graphStruct, bool verbose) {
	node n = graphStruct->nodeCount;
	cout << "** Graph (num node: " << n << ", num edges: " << graphStruct->edgeSize << ")" << endl;
	cout << "** Coloring (num colors: " << col->numOfColors << ")" << endl;
	if (verbose) {
		for (uint i = 1; i <= col->numOfColors; i++) {
			cout << "   color(" << i << ")" << "-> ";
			for (uint j = 0; j < n; j++)
				if (col->coloring[j] == i)
					cout << j << " ";
			cout << "\n";
		}
		cout << "\n";
	}
}

