
#include "Colorer.h"
#include "utils/common.h"
#include "graph/graph.h"
#include "utils/ColoringValidator.h"
#include <iostream>

int main(void) {
	unsigned int n = 10000;		 // 80k ok 115 mln edge
	float prob = .018;				    // density (percentage) for random graphs
	std::default_random_engine engine{ 0 };  // fixed seed

	// new graph with n nodes
	Graph graph(Graph::MemoryEnum::ManagedAllocated);
	//graph.ReadFromMtxFile("inputData/kron_g500-logn21.mtx");
	//GraphStruct* d_GraphStruct;
	//graph.copyToDevice(d_GraphStruct);

	// generate a random graph
	graph.randGraph(prob, engine, n);

	// get the graph struct
	GraphStruct* graphStruct = graph.getStruct();

	printf("start, edgeCount: %d\n", graphStruct->edgeCount);
	printf("start, nodeCount: %d\n", graphStruct->nodeCount);

	// print small graph
	if (n <= 128) {
		//graph.print(true);  // CPU print
		print_d <<<1, 1 >>> (graphStruct, true);  // GPU print
		hipDeviceSynchronize();
	}

	//-------------- START TIME ----------------//
	double start = seconds();

	
	//Colorer colorer(&graph);
	//Coloring* coloring = RandomPriorityColoring(graph);     // 0.375 20k 1.509 no inbound
	//Coloring* coloring = RandomPriorityColoringV2(graph); // 0.352     20k 1.424 con inbounds 0.97 msi
	//Coloring* coloring = RandomPriorityColoringV3(graph); //                                0.96 72 colors
	Coloring* coloring = LDFColoringV3(graph);              //                                     70 colors
	//test(graph);

	double stop = seconds();
	//-------------- END TIME ----------------//

	//printColoring(coloring, graphStruct, 1);
	std::cout << "Iterations: " << coloring->iterationCount << std::endl;

	std::cout << elapsedTime(start, stop) << std::endl;

	validateColoring(coloring, graphStruct);

	return EXIT_SUCCESS;
}
