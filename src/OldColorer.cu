#include "hip/hip_runtime.h"

Coloring* RandomPriorityColoringV2(Graph& graph) // Versione senza dag, solo con inbound count e senza bitmaps
{
	// Alloc and Init returning struct
	Coloring* coloring;
	int n = graph.getStruct()->nodeCount;
	CHECK(hipMallocManaged(&coloring, sizeof(Coloring)));
	CHECK(hipMallocManaged(&(coloring->coloring), n * sizeof(uint)));
	CHECK(hipMallocManaged(&(coloring->coloredNodes), n * sizeof(bool)));
	memset(coloring->coloring, 0, n * sizeof(uint));
	memset(coloring->coloredNodes, 0, n * sizeof(bool));
	coloring->uncoloredFlag = true;
	coloring->iterationCount = 0;
	GraphStruct* graphStruct = graph.getStruct();

	// Generate random node priorities
	hiprandState_t* states;
	uint* priorities;
	hipMalloc((void**)&states, n * sizeof(hiprandState_t));
	hipMalloc((void**)&priorities, n * sizeof(uint));
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);
	uint seed = 0;
	InitRandomPriorities << <gridDim, blockDim >> > (seed, states, priorities, n);
	hipDeviceSynchronize();

	// Calculate inbound counters
	uint* inboundCounts;
	CHECK(hipMalloc((void**)&inboundCounts, n * sizeof(uint)));
	hipMemset(inboundCounts, 0, n * sizeof(uint));
	calculateInbounds << <gridDim, blockDim >> > (graphStruct, inboundCounts, priorities, n);
	hipDeviceSynchronize();

	// Alloc buffer needed to synchronize the coloring
	unsigned* buffer;
	hipMalloc((void**)&buffer, n * sizeof(unsigned));
	hipMemset(buffer, 0, n * sizeof(unsigned));
	bool* filledBuffer;
	hipMalloc((void**)&filledBuffer, n * sizeof(bool));
	hipMemset(filledBuffer, 0, n * sizeof(bool));

	// Color TODO: tieni il flag sulla gpu e itera con gli stream
	coloring->iterationCount = 0;
	while (coloring->uncoloredFlag) {
		coloring->uncoloredFlag = false;
		colorWithInboundCounters << <gridDim, blockDim >> > (coloring, graphStruct, inboundCounts, buffer, filledBuffer);
		hipDeviceSynchronize();
		applyBufferWithInboundCounters << <gridDim, blockDim >> > (coloring, graphStruct, priorities, inboundCounts, buffer, filledBuffer);
		hipDeviceSynchronize();
		coloring->iterationCount++;
	}

	// Free
	hipFree(states);
	hipFree(priorities);
	hipFree(inboundCounts);
	hipFree(buffer);
	hipFree(filledBuffer);
	//hipFree(coloring);
	//hipFree(coloring->coloring);
	//hipFree(coloring->coloredNodes);

	return coloring;

}

Coloring* RandomPriorityColoringV3(Graph& graph) // V2 + bitmaps
{
	// Alloc and Init returning struct
	Coloring* coloring;
	int n = graph.getStruct()->nodeCount;
	CHECK(hipMallocManaged(&coloring, sizeof(Coloring)));
	CHECK(hipMallocManaged(&(coloring->coloring), n * sizeof(uint)));
	CHECK(hipMallocManaged(&(coloring->coloredNodes), n * sizeof(bool)));
	memset(coloring->coloring, 0, n * sizeof(uint));
	memset(coloring->coloredNodes, 0, n * sizeof(bool));
	coloring->uncoloredFlag = true;
	coloring->iterationCount = 0;
	GraphStruct* graphStruct = graph.getStruct();

	// Generate random node priorities
	hiprandState_t* states;
	uint* priorities;
	hipMalloc((void**)&states, n * sizeof(hiprandState_t));
	hipMalloc((void**)&priorities, n * sizeof(uint));
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);
	uint seed = 0;
	InitRandomPriorities << <gridDim, blockDim >> > (seed, states, priorities, n);
	hipDeviceSynchronize();

	// Calculate inbound counters
	uint* inboundCounts;
	CHECK(hipMalloc((void**)&inboundCounts, n * sizeof(uint)));
	hipMemset(inboundCounts, 0, n * sizeof(uint));
	calculateInbounds << <gridDim, blockDim >> > (graphStruct, inboundCounts, priorities, n);
	hipDeviceSynchronize();

	// inizialize bitmaps, every node has a bitmap with a length of inbound edges + 1 TODO: aloc on gpu
	// vision: allocare tutto in un array come al solito ma serve la prefix sum
	// alternativa1: sequenziale O(n)
	// alternativa2: le bitmap vengono allocate staticamente nel kernel, basterebbe poi costruire un index, non sono sequenziali ma penso sia ok
	bool* bitmaps;
	uint bitCount = (n + (int)(graphStruct->edgeCount + 1) / 2);
	CHECK(hipMallocManaged(&(bitmaps), bitCount * sizeof(bool)));
	memset(bitmaps, 1, bitCount * sizeof(bool));
	uint* bitmapIndex;
	CHECK(hipMallocManaged(&bitmapIndex, (n + 1) * sizeof(uint)));
	bitmapIndex[0] = 0;
	uint* h_InboundCounts;
	h_InboundCounts = (uint*)malloc(n * sizeof(uint));
	hipMemcpy(h_InboundCounts, inboundCounts, n * sizeof(uint), hipMemcpyDeviceToHost);
	for (int i = 1; i < n + 1; i++)
		bitmapIndex[i] = bitmapIndex[i - 1] + h_InboundCounts[i - 1] + 1; //the inbound should be only in gpu mem TODO: parallelize with scan

	// Alloc buffer needed to synchronize the coloring
	unsigned* buffer;
	hipMalloc((void**)&buffer, n * sizeof(unsigned));
	hipMemset(buffer, 0, n * sizeof(unsigned));
	bool* filledBuffer;
	hipMalloc((void**)&filledBuffer, n * sizeof(bool));
	hipMemset(filledBuffer, 0, n * sizeof(bool));

	// Color TODO: tieni il flag sulla gpu e itera con gli stream
	coloring->iterationCount = 0;
	while (coloring->uncoloredFlag) {
		coloring->uncoloredFlag = false;
		colorWithInboundCountersBitmaps << <gridDim, blockDim >> > (coloring, graphStruct, inboundCounts, buffer, filledBuffer, bitmaps, bitmapIndex);
		hipDeviceSynchronize();
		applyBufferWithInboundCountersBitmaps << <gridDim, blockDim >> > (coloring, graphStruct, priorities, inboundCounts, buffer, filledBuffer, bitmaps, bitmapIndex);
		hipDeviceSynchronize();
		coloring->iterationCount++;
	}

	// Free
	hipFree(states);
	hipFree(priorities);
	hipFree(inboundCounts);
	hipFree(buffer);
	hipFree(filledBuffer);
	//hipFree(coloring);
	//hipFree(coloring->coloring);
	//hipFree(coloring->coloredNodes);

	return coloring;
}

__global__ void initLDF(GraphStruct* graphStruct, int* inboundCounts, int n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;

	uint degree = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];

	inboundCounts[idx] = 0;
	for (uint i = 0; i < degree; ++i)
	{
		uint neighID = graphStruct->neighs[graphStruct->neighIndex[idx] + i];
		uint neighDegree = graphStruct->neighIndex[neighID + 1] - graphStruct->neighIndex[neighID]; // ottimizzabile su CPU
		if (degree > neighDegree)
		{
			atomicAdd(&inboundCounts[neighID], 1);
			printf(" atomicAdd node(%d -> %d [count: %d] \n", idx, neighID, inboundCounts[neighID]);
		}
		else if (degree == neighDegree && idx > neighID)
		{
			atomicAdd(&inboundCounts[neighID], 1);
			printf(" atomicAdd node(%d -> %d [count: %d] \n", idx, neighID, inboundCounts[neighID]);
		}
	}
}

__global__ void initLDF2(GraphStruct* graphStruct, uint* inboundCounts, int n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;

	uint degree = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];

	inboundCounts[idx] = 0;
	for (uint i = 0; i < degree; ++i) //TODO: ciclo inutile, basta mettere pi� 1 a ogni elemento della lista
	{
		uint neighID = graphStruct->neighs[graphStruct->neighIndex[idx] + i];

		atomicAdd(&inboundCounts[neighID], 1);
		//printf(" atomicAdd node(%d -> %d [count: %d] \n", idx, neighID, inboundCounts[neighID]);

	}
}

Coloring* Colorer::LDFColoring()
{
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((m_GraphStruct->nodeCount + blockDim.x - 1) / blockDim.x, 1, 1);

	// Init DAG TODO: refactorare
	GraphStruct* dag;
	CHECK(hipMallocManaged(&dag, sizeof(GraphStruct)));
	CHECK(hipMallocManaged(&(dag->neighIndex), (m_GraphStruct->nodeCount + 1) * sizeof(int)));
	CHECK(hipMallocManaged(&(dag->neighs), (m_GraphStruct->edgeCount + 1) / 2 * sizeof(int)));
	m_Graph->getLDFDag(dag);

	//initLDF <<<gridDim, blockDim>>> (m_GraphStruct, m_InboundCounts, m_GraphStruct->nodeCount);
	initLDF2 << <gridDim, blockDim >> > (dag, m_InboundCounts, m_GraphStruct->nodeCount);
	hipDeviceSynchronize();

	// inizialize bitmaps
	// Every node has a bitmap with a length of inbound edges + 1
	bool* bitmaps;
	uint bitCount = (m_GraphStruct->nodeCount + (int)(m_GraphStruct->edgeCount + 1) / 2);
	CHECK(hipMallocManaged(&(bitmaps), bitCount * sizeof(bool)));
	memset(bitmaps, 1, bitCount * sizeof(bool));
	uint* bitmapIndex;
	CHECK(hipMallocManaged(&bitmapIndex, (m_GraphStruct->nodeCount + 1) * sizeof(uint)));
	hipDeviceSynchronize();
	bitmapIndex[0] = 0;
	for (int i = 1; i < m_GraphStruct->nodeCount + 1; i++)
		bitmapIndex[i] = bitmapIndex[i - 1] + m_InboundCounts[i - 1] + 1; //this info should be taken by the dag and the inbound should be only in gpu mem

	uint iterationCount = 0;
	while (m_Coloring->uncoloredFlag) {
		m_Coloring->uncoloredFlag = false;
		iterationCount++;
		printf("------------ Sequential iteration: %d \n", iterationCount);
		int deb_inBoundSum = 0;
		for (int i = 0; i < m_GraphStruct->nodeCount; ++i)
		{
			deb_inBoundSum += m_InboundCounts[i];
		}
		printf("------------ inboundsum: %d \n", deb_inBoundSum);
		printf("edges: %d", m_GraphStruct->edgeCount);
		int deb_ready = 0;
		for (int i = 0; i < m_GraphStruct->nodeCount; ++i)
		{
			if (m_InboundCounts[i] == 0 && m_Coloring->coloredNodes[i] == false)
				++deb_ready;
		}
		if (deb_ready == 0)
			printf("------------ ready: %d \n", deb_ready);
		findISLDF << < gridDim, blockDim >> > (m_Coloring, dag, bitmaps, bitmapIndex, m_InboundCounts);
		hipDeviceSynchronize();
	}

	return m_Coloring;
}

void test(Graph& graph)
{
	GraphStruct* graphStruct = graph.getStruct();
	int n = graphStruct->nodeCount;
	uint* h_InboundCounts;
	h_InboundCounts = (uint*)malloc(n * sizeof(uint));

	//priorities
	uint* priorities;
	hipMalloc((void**)&priorities, n * sizeof(uint));
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);
	InitLDFPriorities << <gridDim, blockDim >> > (graphStruct, priorities, n);
	hipDeviceSynchronize();

	//inbounds
	uint* inboundCounts;
	CHECK(hipMalloc((void**)&inboundCounts, n * sizeof(uint)));
	hipMemset(inboundCounts, 0, n * sizeof(uint));
	calculateInbounds << <gridDim, blockDim >> > (graphStruct, inboundCounts, priorities, n);
	hipDeviceSynchronize();

	hipMemcpy(h_InboundCounts, inboundCounts, n * sizeof(uint), hipMemcpyDeviceToHost);

	testAtomicAdd << <gridDim, blockDim >> > (graphStruct, priorities, inboundCounts);
	hipDeviceSynchronize();


	hipMemcpy(h_InboundCounts, inboundCounts, n * sizeof(uint), hipMemcpyDeviceToHost);
	for (int i = 0; i < n; ++i)
	{
		if (h_InboundCounts[0] != 0)
			std::cout << "error" << std::endl;
	}
	std::cout << "end" << std::endl;
}