#include "hip/hip_runtime.h"
﻿
#include ""
#include <iostream>
#include "graph/graph_d.h"
#include "utils/common.h"
#include <hip/hip_cooperative_groups.h>

#include "colorer.h"

#define THREADxBLOCK 128

Colorer::Colorer(Graph* graph)
{
	m_Graph = graph;
	m_GraphStruct = graph->getStruct();
	CHECK(hipMallocManaged(&m_Coloring, sizeof(Coloring)));
	m_Coloring->uncoloredFlag = true;
	m_Coloring->numOfColors = 0;

	uint n = m_GraphStruct->nodeCount;

	CHECK(hipMallocManaged(&m_Coloring->coloring, n * sizeof(uint)));
	memset(m_Coloring->coloring, 0, n * sizeof(uint));
	CHECK(hipMallocManaged(&m_Coloring->coloredNodes, n * sizeof(bool)));
	memset(m_Coloring->coloredNodes, 0, n * sizeof(bool));

	//init inbound counts
	CHECK(hipMallocManaged(&m_InboundCounts, n * sizeof(uint)));
}

Colorer::~Colorer(){
	hipFree(m_InboundCounts);
}

__global__ void initLDF(GraphStruct* graphStruct, int* inboundCounts, int n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;

	uint degree = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];
	printf("node(%d [myDegree: %d] \n", idx, degree);

	inboundCounts[idx] = 0;
	for (uint i = 0; i < degree; ++i)
	{
		uint neighID = graphStruct->neighs[graphStruct->neighIndex[idx] + i];
		uint neighDegree = graphStruct->neighIndex[neighID + 1] - graphStruct->neighIndex[neighID]; // ottimizzabile su CPU
		if (degree > neighDegree)
		{
			atomicAdd(&inboundCounts[neighID], 1);
			printf(" atomicAdd node(%d -> %d [count: %d] \n", idx, neighID, inboundCounts[neighID]);
		}
		else if (degree == neighDegree && idx > neighID)
		{
			atomicAdd(&inboundCounts[neighID], 1);
			printf(" atomicAdd node(%d -> %d [count: %d] \n", idx, neighID, inboundCounts[neighID]);
		}
	}
}

__global__ void initLDF2(GraphStruct* graphStruct, uint* inboundCounts, int n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;

	uint degree = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];
	//printf("node(%d [myDegree: %d] \n", idx, degree);

	inboundCounts[idx] = 0;
	for (uint i = 0; i < degree; ++i) //TODO: ciclo inutile, basta mettere piú 1 a ogni elemento della lista
	{
		uint neighID = graphStruct->neighs[graphStruct->neighIndex[idx] + i];
		
		atomicAdd(&inboundCounts[neighID], 1);
		//printf(" atomicAdd node(%d -> %d [count: %d] \n", idx, neighID, inboundCounts[neighID]);
		
	}
}

__global__ void findISLDF(Coloring* coloring, GraphStruct* graphStruct, bool* bitmaps, uint* bitmapIndex, uint* inboundCounts)
{
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= graphStruct->nodeCount) //é giusto
		return;

	if (coloring->coloredNodes[idx])
		return;

	//printf("GPU - I'm %d, myInbound: %d\n", idx, inboundCounts[idx]);

	uint offset = graphStruct->neighIndex[idx];
	uint deg = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];

	if (inboundCounts[idx] == 0) // Ready node
	{
		int colorCount = bitmapIndex[idx + 1] - bitmapIndex[idx];
		printf("I'm %d, total colors: %d\n", idx, colorCount);

		int bestColor = colorCount;
		for (int i = 0; i < colorCount; ++i)
		{
			if (bitmaps[bitmapIndex[idx] + i])
			{
					bestColor = i;
					break;
			}
		}
		coloring->coloring[idx] = bestColor;
		coloring->coloredNodes[idx] = true;
		printf("colored: %d, best color: %d: \n", idx, coloring->coloring[idx]);
		
		for (uint i = 0; i < deg; i++) {
			uint neighID = graphStruct->neighs[offset + i];
			if (!coloring->coloredNodes[neighID])
			{
				atomicAdd(&inboundCounts[neighID], -1);
				bitmaps[bitmapIndex[neighID] + bestColor] = 0;
			}
		}
	}
	else
	{
		coloring->uncoloredFlag = true;
	}
}



Coloring* Colorer::LDFColoring()
{
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((m_GraphStruct->nodeCount + blockDim.x - 1) / blockDim.x, 1, 1);
	
	// Init DAG TODO: refactorare
	GraphStruct* dag;
	CHECK(hipMallocManaged(&dag, sizeof(GraphStruct)));
	CHECK(hipMallocManaged(&(dag->neighIndex), (m_GraphStruct->nodeCount + 1) * sizeof(int)));
	CHECK(hipMallocManaged(&(dag->neighs), (m_GraphStruct->edgeCount+1)/2 * sizeof(int)));
	m_Graph->getLDFDag(dag);

	//initLDF <<<gridDim, blockDim>>> (m_GraphStruct, m_InboundCounts, m_GraphStruct->nodeCount);
	initLDF2 <<<gridDim, blockDim>>> (dag, m_InboundCounts, m_GraphStruct->nodeCount);
	hipDeviceSynchronize();

	// inizialize bitmaps
	// Every node has a bitmap with a length of inbound edges + 1
	bool* bitmaps;
	uint bitCount = (m_GraphStruct->nodeCount + (int)(m_GraphStruct->edgeCount + 1) / 2);
	CHECK(hipMallocManaged(&(bitmaps), bitCount * sizeof(bool)));
	memset(bitmaps, 1, bitCount * sizeof(bool));
	uint* bitmapIndex;
	CHECK(hipMallocManaged(&bitmapIndex, (m_GraphStruct->nodeCount + 1) * sizeof(uint)));
	hipDeviceSynchronize();
	bitmapIndex[0] = 0;
	for (int i = 1; i < m_GraphStruct->nodeCount + 1; i++)
		bitmapIndex[i] = bitmapIndex[i - 1] + m_InboundCounts[i - 1] + 1; //this info should be taken by the dag and the inbound should be only in gpu mem

	uint iterationCount = 0;
	while (m_Coloring->uncoloredFlag) {
		m_Coloring->uncoloredFlag = false;
		iterationCount++;
		printf("------------ Sequential iteration: %d \n", iterationCount);
		int deb_inBoundSum = 0;
		for (int i = 0; i < m_GraphStruct->nodeCount; ++i)
		{
			deb_inBoundSum += m_InboundCounts[i];
		}
		printf("------------ inboundsum: %d \n", deb_inBoundSum);
		printf("edges: %d", m_GraphStruct->edgeCount);
		int deb_ready = 0;
		for (int i = 0; i < m_GraphStruct->nodeCount; ++i)
		{
			if (m_InboundCounts[i] == 0 && m_Coloring->coloredNodes[i] == false)
				++deb_ready;
		}
		if (deb_ready == 0)
			printf("------------ ready: %d \n", deb_ready);
		findISLDF <<< gridDim, blockDim >>> (m_Coloring, dag, bitmaps, bitmapIndex, m_InboundCounts);
		hipDeviceSynchronize();
	}

	return m_Coloring;
}

Coloring* Colorer::RandomPriorityColoringCPUSequential()
{
	// DAG
	Graph dag(Graph::MemoryEnum::HostAllocated);
	m_Graph->BuildRandomDAG(dag);

	// temp data inizialization
	uint bitCount = (m_GraphStruct->nodeCount + (int)(m_GraphStruct->edgeCount + 1) / 2);
	std::vector<bool> bitmaps(bitCount, true);
	std::vector<uint> bitmapIndex(m_GraphStruct->nodeCount + 1);
	std::vector<uint> inboundCounts(m_GraphStruct->nodeCount, 0);
	GraphStruct* dagStruct = dag.getStruct();
	for (int i = 0; i < dag.GetEdgeCount(); ++i)
		inboundCounts[dagStruct->neighs[i]]++;
	for (int i = 1; i < m_GraphStruct->nodeCount + 1; i++)
		bitmapIndex[i] = bitmapIndex[i - 1] + m_InboundCounts[i - 1] + 1;

	// JP Coloring
	m_Coloring->numOfColors = 0;
	while (m_Coloring->uncoloredFlag)
	{
		m_Coloring->uncoloredFlag = false;
		for (int i = 0; i < m_GraphStruct->nodeCount; ++i)
		{
			if (m_Coloring->coloring[i])
				continue;

			uint offset = dagStruct->neighIndex[i];
			uint deg = dagStruct->neighIndex[i + 1] - dagStruct->neighIndex[i];

			if (inboundCounts[i] == 0) // Ready node
			{
				int colorCount = bitmapIndex[i + 1] - bitmapIndex[i];
				printf("I'm %d, total colors: %d\n", i, colorCount);

				int bestColor = colorCount;
				for (int j = 0; j < colorCount; ++j)
				{
					if (bitmaps[bitmapIndex[i] + j])
					{
						if (j < bestColor)
						{
							//TODO: find another way
							bestColor = j;
							break;
						}
					}
				}
				m_Coloring->coloring[i] = bestColor;
				m_Coloring->coloredNodes[i] = true;
				printf("colored: %d, best color: %d: \n", i, m_Coloring->coloring[i]);
				if (bestColor > m_Coloring->numOfColors)
				{
					m_Coloring->numOfColors = bestColor; // possibile race, potrei computarlo nella print
				}
				for (uint j = 0; j < deg; j++) {
					uint neighID = dagStruct->neighs[offset + j];
					inboundCounts[neighID]--;
					bitmaps[bitmapIndex[neighID] + bestColor] = 0;
					
				}
			}
			else
			{
				m_Coloring->uncoloredFlag = true;
			}
		}
	}
	return m_Coloring;
}

void mallocOnHost(Coloring* coloring, unsigned n)
{
	coloring = (Coloring*)malloc(sizeof(Coloring));
	coloring->coloring = (uint*)calloc(n, sizeof(uint));
	coloring->coloredNodes = (bool*)calloc(n, sizeof(bool));
}

//TODO: evita atomic add facendo il confronto al contrario
__global__ void calculateInbounds(GraphStruct* graphStruct, unsigned int* inboundCounts, unsigned int* priorities, int n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;

	uint degree = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];
	//printf("node(%d [myDegree: %d] \n", idx, degree);

	inboundCounts[idx] = 0;
	for (uint i = 0; i < degree; ++i)
	{
		uint neighID = graphStruct->neighs[graphStruct->neighIndex[idx] + i];
		if (priorities[idx] > priorities[neighID])
		{
			atomicAdd(&inboundCounts[neighID], 1);
			//printf(" atomicAdd node(%d -> %d [count: %d] \n", idx, neighID, inboundCounts[neighID]);
		}
		else if (priorities[idx] == priorities[neighID] && idx > neighID)
		{
			atomicAdd(&inboundCounts[neighID], 1);
			//printf(" atomicAdd node(%d -> %d [count: %d] \n", idx, neighID, inboundCounts[neighID]);
		}
	}
}





__global__ void buildParallelDag()
{

}





/**
 * find an IS
 */
__global__ void findIS(Coloring* col, GraphStruct* graphStruct, uint* weights, unsigned* buffer, bool* filledBuffer)
{
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= graphStruct->nodeCount)
		return;

	if (col->coloredNodes[idx])
		return;

	uint offset = graphStruct->neighIndex[idx];
	uint deg = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];

	bool candidate = true;
	for (uint j = 0; j < deg; j++) {
		uint neighID = graphStruct->neighs[offset + j];

		if (!col->coloredNodes[neighID] &&
			((weights[idx] < weights[neighID]) || ((weights[idx] == weights[neighID]) && idx < neighID))) {
			candidate = false;
		}
	}
	if (candidate) {
		buffer[idx] = col->numOfColors;
		filledBuffer[idx] = true;
		//printf("candidate: %d, color: %d\n", idx, col->numOfColors);
	}
	else
	{
		col->uncoloredFlag = true;
		//printf("not candidate: %d, color: %d\n", idx, col->numOfColors);
	}
}

__global__ void applyBuffer(Coloring* coloring, unsigned* buffer, bool* filledBuffer, unsigned n)
{
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= n)
		return;

	if (coloring->coloredNodes[idx])
		return;

	if (!filledBuffer[idx])
		return;

	coloring->coloring[idx] = buffer[idx];
	coloring->coloredNodes[idx] = true;
	filledBuffer[idx] = false;
	//printf("buffer applied: %d, color: %d\n", idx, coloring->coloring[idx]);

}

__global__ void applyBufferWithInboundCounters(Coloring* coloring, GraphStruct* graphStruct, unsigned* priorities, unsigned* inboundCounts,unsigned* buffer, bool* filledBuffer)
{
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= graphStruct->nodeCount)
		return;

	if (coloring->coloredNodes[idx])
		return;

	if (!filledBuffer[idx])
		return;

	uint offset = graphStruct->neighIndex[idx];
	uint deg = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];

	for (uint i = 0; i < deg; i++)
	{
		//TODO: check if there is arc in the dag, it could cause troubles when I will implement shortcuts
		uint neighID = graphStruct->neighs[offset + i];
		if (!coloring->coloredNodes[neighID] &&
			((priorities[idx] > priorities[neighID]) || ((priorities[idx] == priorities[neighID]) && idx > neighID)))
		{
			atomicAdd(&inboundCounts[neighID], -1);
			//if (neighID == 750)
			//	printf("I'm: %d, removed arc to: %d: \n", idx, neighID);
		}
		//printf("I'm: %d, removed arc to: %d: \n", idx, neighID);
		
	}
	coloring->coloring[idx] = buffer[idx];
	coloring->coloredNodes[idx] = true;
	filledBuffer[idx] = false;
	//printf("buffer applied: %d, color: %d\n", idx, coloring->coloring[idx]);

}

__global__ void colorWithInboundCounters(Coloring* coloring, GraphStruct* graphStruct, uint* inboundCounts, uint* buffer, bool* filledBuffer)
{
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= graphStruct->nodeCount)
		return;

	if (coloring->coloredNodes[idx])
		return;

	//printf("GPU - I'm %d, myInbound: %d\n", idx, inboundCounts[idx]);

	

	//if (idx == 750)
	//	printf("uncolored: %d, still: %d -- best color: %d: \n", idx, inboundCounts[idx], coloring->numOfColors);
	if (inboundCounts[idx] == 0) // Ready node
	{
		buffer[idx] = coloring->numOfColors;
		filledBuffer[idx] = true;
	}
	else
	{
		coloring->uncoloredFlag = true;
	}
}



/**
 *  this GPU kernel takes an array of states, and an array of ints, and puts a random int into each
 */
__global__ void InitRandomPriorities(uint seed, hiprandState_t* states, uint* priorities, uint n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;
	hiprand_init(seed, idx, 0, &states[idx]);
	priorities[idx] = hiprand(&states[idx]) % n * n;
}

__global__ void InitLDFPriorities(GraphStruct* graphStruct, uint* priorities, uint n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;
	priorities[idx] = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];
}



/**
 * Print the graph (verbose = 1 for "verbose print")
 * @param verbose print the complete graph
 */
void printColoring(Coloring* col, GraphStruct* graphStruct, bool verbose) {
	unsigned n = graphStruct->nodeCount;
	std::cout << "** Graph (num node: " << n << ", num edges: " << graphStruct->edgeCount << ")" << std::endl;
	std::cout << "** Coloring (num colors: " << col->numOfColors + 1 << ")" << std::endl;
	if (verbose) {
		for (uint i = 0; i <= col->numOfColors; i++) {
			std::cout << "   color(" << i << ")" << "-> ";
			for (uint j = 0; j < n; j++)
				if (col->coloring[j] == i)
					std::cout << j << " ";
			std::cout << "\n";
		}
		std::cout << "\n";
	}
}

Coloring* RandomPriorityColoring(Graph& graph) // no inboundsCount, no bitmap no dag
{
	// Alloc and Init returning struct
	Coloring* coloring;
	int n = graph.getStruct()->nodeCount;
	CHECK(hipMallocManaged(&coloring, sizeof(Coloring)));
	CHECK(hipMallocManaged(&(coloring->coloring), n * sizeof(uint)));
	CHECK(hipMallocManaged(&(coloring->coloredNodes), n * sizeof(bool)));
	memset(coloring->coloring, 0, n * sizeof(uint));
	memset(coloring->coloredNodes, 0, n * sizeof(bool));
	coloring->uncoloredFlag = true;
	coloring->numOfColors = 0;
	GraphStruct* graphStruct = graph.getStruct();

	// Generate random node priorities
	hiprandState_t* states;
	uint* priorities;
	hipMalloc((void**)&states, n * sizeof(hiprandState_t));
	hipMalloc((void**)&priorities, n * sizeof(uint));
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);
	uint seed = 0;
	InitRandomPriorities << <gridDim, blockDim >> > (seed, states, priorities, n);
	hipDeviceSynchronize();

	// Alloc buffer needed to synchronize the coloring
	unsigned* buffer;
	hipMalloc((void**)&buffer, n * sizeof(unsigned));
	hipMemset(buffer, 0, n * sizeof(unsigned));
	bool* filledBuffer;
	hipMalloc((void**)&filledBuffer, n * sizeof(bool));
	hipMemset(filledBuffer, 0, n * sizeof(bool));

	// Color TODO: tieni il flag sulla gpu e itera con gli stream
	coloring->numOfColors = 0;
	while (coloring->uncoloredFlag) {
		coloring->uncoloredFlag = false;
		findIS << <gridDim, blockDim >> > (coloring, graphStruct, priorities, buffer, filledBuffer);
		hipDeviceSynchronize();
		applyBuffer << <gridDim, blockDim >> > (coloring, buffer, filledBuffer, n);
		hipDeviceSynchronize();
		coloring->numOfColors++;
	}

	// Free
	hipFree(states);
	hipFree(priorities);
	hipFree(buffer);
	hipFree(filledBuffer);
	//hipFree(coloring);
	//hipFree(coloring->coloring);
	//hipFree(coloring->coloredNodes);

	return coloring;
}

Coloring* RandomPriorityColoringV2(Graph& graph) // Versione senza dag, solo con inbound count e senza bitmaps
{
	// Alloc and Init returning struct
	Coloring* coloring;
	int n = graph.getStruct()->nodeCount;
	CHECK(hipMallocManaged(&coloring, sizeof(Coloring)));
	CHECK(hipMallocManaged(&(coloring->coloring), n * sizeof(uint)));
	CHECK(hipMallocManaged(&(coloring->coloredNodes), n * sizeof(bool)));
	memset(coloring->coloring, 0, n * sizeof(uint));
	memset(coloring->coloredNodes, 0, n * sizeof(bool));
	coloring->uncoloredFlag = true;
	coloring->numOfColors = 0;
	GraphStruct* graphStruct = graph.getStruct();

	// Generate random node priorities
	hiprandState_t* states;
	uint* priorities;
	hipMalloc((void**)&states, n * sizeof(hiprandState_t));
	hipMalloc((void**)&priorities, n * sizeof(uint));
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);
	uint seed = 0;
	InitRandomPriorities << <gridDim, blockDim >> > (seed, states, priorities, n);
	hipDeviceSynchronize();

	// Calculate inbound counters
	uint* inboundCounts;
	CHECK(hipMalloc((void**)&inboundCounts, n * sizeof(uint)));
	hipMemset(inboundCounts, 0, n * sizeof(uint));
	calculateInbounds << <gridDim, blockDim >> > (graphStruct, inboundCounts, priorities, n);
	hipDeviceSynchronize();

	// Alloc buffer needed to synchronize the coloring
	unsigned* buffer;
	hipMalloc((void**)&buffer, n * sizeof(unsigned));
	hipMemset(buffer, 0, n * sizeof(unsigned));
	bool* filledBuffer;
	hipMalloc((void**)&filledBuffer, n * sizeof(bool));
	hipMemset(filledBuffer, 0, n * sizeof(bool));

	// Color TODO: tieni il flag sulla gpu e itera con gli stream
	coloring->numOfColors = 0;
	while (coloring->uncoloredFlag) {
		coloring->uncoloredFlag = false;
		colorWithInboundCounters <<<gridDim, blockDim>>> (coloring, graphStruct, inboundCounts, buffer, filledBuffer);
		hipDeviceSynchronize();
		applyBufferWithInboundCounters <<<gridDim, blockDim>>>(coloring, graphStruct, priorities, inboundCounts, buffer, filledBuffer);
		hipDeviceSynchronize();
		coloring->numOfColors++;
	}

	// Free
	hipFree(states);
	hipFree(priorities);
	hipFree(inboundCounts);
	hipFree(buffer);
	hipFree(filledBuffer);
	//hipFree(coloring);
	//hipFree(coloring->coloring);
	//hipFree(coloring->coloredNodes);

	return coloring;

}

Coloring* RandomPriorityColoringV3(Graph& graph) // V2 + bitmaps
{
	// Alloc and Init returning struct
	Coloring* coloring;
	int n = graph.getStruct()->nodeCount;
	CHECK(hipMallocManaged(&coloring, sizeof(Coloring)));
	CHECK(hipMallocManaged(&(coloring->coloring), n * sizeof(uint)));
	CHECK(hipMallocManaged(&(coloring->coloredNodes), n * sizeof(bool)));
	memset(coloring->coloring, 0, n * sizeof(uint));
	memset(coloring->coloredNodes, 0, n * sizeof(bool));
	coloring->uncoloredFlag = true;
	coloring->numOfColors = 0;
	GraphStruct* graphStruct = graph.getStruct();

	// Generate random node priorities
	hiprandState_t* states;
	uint* priorities;
	hipMalloc((void**)&states, n * sizeof(hiprandState_t));
	hipMalloc((void**)&priorities, n * sizeof(uint));
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);
	uint seed = 0;
	InitRandomPriorities <<<gridDim, blockDim >> > (seed, states, priorities, n);
	hipDeviceSynchronize();

	// Calculate inbound counters
	uint* inboundCounts;
	CHECK(hipMalloc((void**)&inboundCounts, n * sizeof(uint)));
	hipMemset(inboundCounts, 0, n * sizeof(uint));
	calculateInbounds << <gridDim, blockDim >> > (graphStruct, inboundCounts, priorities, n);
	hipDeviceSynchronize();

	// inizialize bitmaps, every node has a bitmap with a length of inbound edges + 1 TODO: aloc on gpu
	// vision: allocare tutto in un array come al solito ma serve la prefix sum
	// alternativa1: sequenziale O(n)
	// alternativa2: le bitmap vengono allocate staticamente nel kernel, basterebbe poi costruire un index, non sono sequenziali ma penso sia ok
	bool* bitmaps;
	uint bitCount = (n + (int)(graphStruct->edgeCount + 1) / 2);
	CHECK(hipMallocManaged(&(bitmaps), bitCount * sizeof(bool)));
	memset(bitmaps, 1, bitCount * sizeof(bool));
	uint* bitmapIndex;
	CHECK(hipMallocManaged(&bitmapIndex, (n + 1) * sizeof(uint)));
	bitmapIndex[0] = 0;
	uint* h_InboundCounts;
	h_InboundCounts = (uint*)malloc(n * sizeof(uint));
	hipMemcpy(h_InboundCounts, inboundCounts, n * sizeof(uint), hipMemcpyDeviceToHost);
	for (int i = 1; i < n + 1; i++)
		bitmapIndex[i] = bitmapIndex[i - 1] + h_InboundCounts[i - 1] + 1; //the inbound should be only in gpu mem TODO: parallelize with scan

	// Alloc buffer needed to synchronize the coloring
	unsigned* buffer;
	hipMalloc((void**)&buffer, n * sizeof(unsigned));
	hipMemset(buffer, 0, n * sizeof(unsigned));
	bool* filledBuffer;
	hipMalloc((void**)&filledBuffer, n * sizeof(bool));
	hipMemset(filledBuffer, 0, n * sizeof(bool));

	// Color TODO: tieni il flag sulla gpu e itera con gli stream
	coloring->numOfColors = 0;
	while (coloring->uncoloredFlag) {
		coloring->uncoloredFlag = false;
		colorWithInboundCountersBitmaps <<<gridDim, blockDim>>> (coloring, graphStruct, inboundCounts, buffer, filledBuffer, bitmaps, bitmapIndex);
		hipDeviceSynchronize();
		applyBufferWithInboundCountersBitmaps <<<gridDim, blockDim>>>(coloring, graphStruct, priorities, inboundCounts, buffer, filledBuffer, bitmaps, bitmapIndex);
		hipDeviceSynchronize();
		coloring->numOfColors++;
	}

	// Free
	hipFree(states);
	hipFree(priorities);
	hipFree(inboundCounts);
	hipFree(buffer);
	hipFree(filledBuffer);
	//hipFree(coloring);
	//hipFree(coloring->coloring);
	//hipFree(coloring->coloredNodes);

	return coloring;
}

Coloring* LDFColoringV3(Graph& graph)
{
	// Alloc and Init returning struct
	Coloring* coloring;
	int n = graph.getStruct()->nodeCount;
	CHECK(hipMallocManaged(&coloring, sizeof(Coloring)));
	CHECK(hipMallocManaged(&(coloring->coloring), n * sizeof(uint)));
	CHECK(hipMallocManaged(&(coloring->coloredNodes), n * sizeof(bool)));
	memset(coloring->coloring, 0, n * sizeof(uint));
	memset(coloring->coloredNodes, 0, n * sizeof(bool));
	coloring->uncoloredFlag = true;
	coloring->numOfColors = 0;
	GraphStruct* graphStruct = graph.getStruct();

	// Generate LDF priorities
	uint* priorities;
	hipMalloc((void**)&priorities, n * sizeof(uint));
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);
	InitLDFPriorities <<<gridDim, blockDim >>> (graphStruct, priorities, n);
	hipDeviceSynchronize();

	// Calculate inbound counters
	uint* inboundCounts;
	CHECK(hipMalloc((void**)&inboundCounts, n * sizeof(uint)));
	hipMemset(inboundCounts, 0, n * sizeof(uint));
	calculateInbounds << <gridDim, blockDim >> > (graphStruct, inboundCounts, priorities, n);
	hipDeviceSynchronize();

	// inizialize bitmaps, every node has a bitmap with a length of inbound edges + 1 TODO: aloc on gpu
	// vision: allocare tutto in un array come al solito ma serve la prefix sum
	// alternativa1: sequenziale O(n)
	// alternativa2: le bitmap vengono allocate staticamente nel kernel, basterebbe poi costruire un index, non sono sequenziali ma penso sia ok
	bool* bitmaps;
	uint bitCount = (n + (int)(graphStruct->edgeCount + 1) / 2);
	CHECK(hipMallocManaged(&(bitmaps), bitCount * sizeof(bool)));
	memset(bitmaps, 1, bitCount * sizeof(bool));
	uint* bitmapIndex;
	CHECK(hipMallocManaged(&bitmapIndex, (n + 1) * sizeof(uint)));
	bitmapIndex[0] = 0;
	uint* h_InboundCounts;
	h_InboundCounts = (uint*)malloc(n * sizeof(uint));
	hipMemcpy(h_InboundCounts, inboundCounts, n * sizeof(uint), hipMemcpyDeviceToHost);
	for (int i = 1; i < n + 1; i++)
		bitmapIndex[i] = bitmapIndex[i - 1] + h_InboundCounts[i - 1] + 1; //the inbound should be only in gpu mem TODO: parallelize with scan

	// Alloc buffer needed to synchronize the coloring
	unsigned* buffer;
	hipMalloc((void**)&buffer, n * sizeof(unsigned));
	hipMemset(buffer, 0, n * sizeof(unsigned));
	bool* filledBuffer;
	hipMalloc((void**)&filledBuffer, n * sizeof(bool));
	hipMemset(filledBuffer, 0, n * sizeof(bool));

	// DEBUG
	uint* h_priorities = (uint*)malloc(n * sizeof(uint));

	// Color TODO: tieni il flag sulla gpu e itera con gli stream
	coloring->numOfColors = 0;
	while (coloring->uncoloredFlag) {
		coloring->uncoloredFlag = false;
		colorWithInboundCountersBitmaps <<<gridDim, blockDim>>> (coloring, graphStruct, inboundCounts, buffer, filledBuffer, bitmaps, bitmapIndex);
		hipDeviceSynchronize();
		applyBufferWithInboundCountersBitmaps <<<gridDim, blockDim>>>(coloring, graphStruct, priorities, inboundCounts, buffer, filledBuffer, bitmaps, bitmapIndex);
		hipDeviceSynchronize();
		coloring->numOfColors++;
		hipMemcpy(h_priorities, priorities, n * sizeof(uint), hipMemcpyDeviceToHost); //TODO: remove
		hipDeviceSynchronize();
	}

	// Free
	hipFree(priorities);
	hipFree(inboundCounts);
	hipFree(buffer);
	hipFree(filledBuffer);
	//hipFree(coloring);
	//hipFree(coloring->coloring);
	//hipFree(coloring->coloredNodes);

	return coloring;
}

__global__ void colorWithInboundCountersBitmaps(Coloring* coloring, GraphStruct* graphStruct, uint* inboundCounts, uint* buffer, bool* filledBuffer, bool* bitmaps, uint* bitmapIndex)
{
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= graphStruct->nodeCount)
		return;

	if (coloring->coloredNodes[idx])
		return;

	//printf("GPU - I'm %d, myInbound: %d\n", idx, inboundCounts[idx]);

	if (idx == 8984)
		printf("uncolored: %d, still: %d -- iteration: %d: \n", idx, inboundCounts[idx], coloring->numOfColors);
	if (inboundCounts[idx] == 0) // Ready node
	{
		int colorCount = bitmapIndex[idx + 1] - bitmapIndex[idx];
		//printf("I'm %d, total colors: %d\n", idx, colorCount);

		int bestColor = colorCount;
		for (int i = 0; i < colorCount; ++i)
		{
			if (bitmaps[bitmapIndex[idx] + i])
			{
				bestColor = i;
				break;
			}
		}
		buffer[idx] = bestColor;
		filledBuffer[idx] = true;
		//printf("I'm %d, filled buffer: %d\n", idx, bestColor);
	}
	else
	{
		coloring->uncoloredFlag = true;
	}
}

__global__ void applyBufferWithInboundCountersBitmaps(Coloring* coloring, GraphStruct* graphStruct, unsigned* priorities, unsigned* inboundCounts, unsigned* buffer, bool* filledBuffer, bool* bitmaps, uint* bitmapIndex)
{
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= graphStruct->nodeCount)
		return;

	if (coloring->coloredNodes[idx])
		return;

	if (!filledBuffer[idx])
		return;

	uint offset = graphStruct->neighIndex[idx];
	uint deg = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];

	for (uint i = 0; i < deg; i++)
	{
		uint neighID = graphStruct->neighs[offset + i];
		
		if (priorities[idx] > priorities[neighID])
		{
			atomicAdd(&inboundCounts[neighID], -1);
			bitmaps[bitmapIndex[neighID] + buffer[idx]] = 0;

			if (neighID == 8984)
				printf("I'm: %d, removed arc to: %d: \n", idx, neighID);
			//printf("I'm: %d, removed arc to: %d: \n", idx, neighID);
		}
		else if (priorities[idx] == priorities[neighID] && idx > neighID)
		{
			atomicAdd(&inboundCounts[neighID], -1);
			bitmaps[bitmapIndex[neighID] + buffer[idx]] = 0;

			if (neighID == 8984)
				printf("I'm: %d, removed arc to: %d: \n", idx, neighID);
			//printf("I'm: %d, removed arc to: %d: \n", idx, neighID);
		}

	}
	coloring->coloring[idx] = buffer[idx];
	coloring->coloredNodes[idx] = true;
	filledBuffer[idx] = false;
	//printf("buffer applied: %d, color: %d\n", idx, coloring->coloring[idx]);
}

__global__ void testAtomicAdd(GraphStruct* graphStruct, unsigned* priorities, unsigned* inboundCounts)
{
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= graphStruct->nodeCount)
		return;

	uint offset = graphStruct->neighIndex[idx];
	uint deg = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];

	for (uint i = 0; i < deg; i++)
	{
		uint neighID = graphStruct->neighs[offset + i];

		if (priorities[idx] > priorities[neighID])
		{
			atomicAdd(&inboundCounts[neighID], -1);

			if (neighID == 8984)
				printf("I'm: %d, removed arc to: %d: \n", idx, neighID);
			//printf("I'm: %d, removed arc to: %d: \n", idx, neighID);
		}
		else if (priorities[idx] == priorities[neighID] && idx > neighID)
		{
			atomicAdd(&inboundCounts[neighID], -1);

			//if (neighID == 8984)
			//	printf("I'm: %d, removed arc to: %d: \n", idx, neighID);
			//printf("I'm: %d, removed arc to: %d: \n", idx, neighID);
		}

	}

}

void test(Graph& graph)
{
	GraphStruct* graphStruct = graph.getStruct();
	int n = graphStruct->nodeCount;
	uint* h_InboundCounts;
	h_InboundCounts = (uint*)malloc(n * sizeof(uint));

	//priorities
	uint* priorities;
	hipMalloc((void**)&priorities, n * sizeof(uint));
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);
	InitLDFPriorities << <gridDim, blockDim >> > (graphStruct, priorities, n);
	hipDeviceSynchronize();

	//inbounds
	uint* inboundCounts;
	CHECK(hipMalloc((void**)&inboundCounts, n * sizeof(uint)));
	hipMemset(inboundCounts, 0, n * sizeof(uint));
	calculateInbounds << <gridDim, blockDim >> > (graphStruct, inboundCounts, priorities, n);
	hipDeviceSynchronize();

	hipMemcpy(h_InboundCounts, inboundCounts, n * sizeof(uint), hipMemcpyDeviceToHost);

	testAtomicAdd << <gridDim, blockDim >> > (graphStruct, priorities, inboundCounts);
	hipDeviceSynchronize();

	
	hipMemcpy(h_InboundCounts, inboundCounts, n * sizeof(uint), hipMemcpyDeviceToHost);
	for (int i = 0; i < n; ++i)
	{
		if (h_InboundCounts[0] != 0)
			std::cout << "error" << std::endl;
	}
	std::cout << "end" << std::endl;
}



