#include "hip/hip_runtime.h"
﻿
#include ""
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include "Colorer.h"

__global__ void calculateInbounds(GraphStruct* graphStruct, unsigned int* inboundCounts, unsigned int* priorities, int n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;

	uint degree = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];
	for (uint i = 0; i < degree; ++i)
	{
		uint neighID = graphStruct->neighs[graphStruct->neighIndex[idx] + i];
		if ((priorities[idx] < priorities[neighID]) ||
				(priorities[idx] == priorities[neighID] && idx < neighID))
			inboundCounts[idx] += 1;
	}
}

__global__ void colorWithInboundCountersBitmaps(uint* coloring, bool* coloredNodes, GraphStruct* graphStruct, uint* inboundCounts, uint* buffer, bool* filledBuffer, bool* bitmaps, uint* bitmapIndex, bool* uncoloredFlag)
{
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= graphStruct->nodeCount)
		return;
	if (coloredNodes[idx])
		return;

	if (inboundCounts[idx] <= 0) // Ready node
	{
		int colorCount = bitmapIndex[idx + 1] - bitmapIndex[idx];

		int bestColor = colorCount;
		for (int i = 0; i < colorCount; ++i)
		{
			if (bitmaps[bitmapIndex[idx] + i])
			{
				bestColor = i;
				break;
			}
		}
		buffer[idx] = bestColor;
		filledBuffer[idx] = true;
	}
	else
	{
		*uncoloredFlag = true;
	}
}

__global__ void applyBufferWithInboundCountersBitmaps(uint* coloring, bool* coloredNodes, GraphStruct* graphStruct, unsigned* priorities, unsigned* inboundCounts, unsigned* buffer, bool* filledBuffer, bool* bitmaps, uint* bitmapIndex)
{
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= graphStruct->nodeCount)
		return;

	if (coloredNodes[idx])
		return;

	if (!filledBuffer[idx])
		return;

	uint offset = graphStruct->neighIndex[idx];
	uint deg = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];

	for (uint i = 0; i < deg; i++)
	{
		uint neighID = graphStruct->neighs[offset + i];

		if ((priorities[idx] > priorities[neighID]) ||
				(priorities[idx] == priorities[neighID] && idx > neighID))
		{
			atomicAdd(&inboundCounts[neighID], -1);
			int colorCount = bitmapIndex[neighID + 1] - bitmapIndex[neighID];
			if (buffer[idx] < colorCount)
				bitmaps[bitmapIndex[neighID] + buffer[idx]] = 0;
		}
	}

	coloring[idx] = buffer[idx];
	coloredNodes[idx] = true;
	filledBuffer[idx] = false;
}

__global__ void colorWithoutInbounds(bool* isColored, GraphStruct* graphStruct, uint* buffer, bool* filledBuffer, bool* bitmaps, uint* bitmapIndex, uint* priorities, bool* uncoloredFlag)
{
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= graphStruct->nodeCount)
		return;
	if (isColored[idx])
		return;

	uint offset = graphStruct->neighIndex[idx];
	uint deg = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];

	bool candidate = true;
	for (uint j = 0; j < deg; j++) {
		uint neighID = graphStruct->neighs[offset + j];

		if (!isColored[neighID] &&
			((priorities[idx] < priorities[neighID]) || ((priorities[idx] == priorities[neighID]) && idx < neighID))) {
			candidate = false;
		}
	}
	if (candidate) {
		int colorCount = bitmapIndex[idx + 1] - bitmapIndex[idx];
		int bestColor = 0;
		for (int i = 0; i < colorCount; ++i)
		{
			if (bitmaps[bitmapIndex[idx] + i])
			{
				bestColor = i;
				break;
			}
		}
		buffer[idx] = bestColor;
		filledBuffer[idx] = true;
	}
	else
	{
		*uncoloredFlag = true;
	}
}

Coloring* global::color(Graph& graph, uint* d_priorities)
{
	// Init
	int n = graph.GetNodeCount();
	int edgeCount = graph.GetEdgeCount();
	GraphStruct* d_graphStruct;
	graph.getDeviceStruct(d_graphStruct);

	// coloring struct init
	uint* coloring = (uint*)malloc(n * sizeof(uint));
	bool* coloredNodes = (bool*)malloc(n * sizeof(bool));
	memset(coloring, 0, n * sizeof(uint));
	memset(coloredNodes, 0, n * sizeof(bool));
	uint* d_coloring;
	bool* d_coloredNodes;
	CHECK(hipMalloc((void**)&(d_coloring), n * sizeof(uint)));
	CHECK(hipMalloc((void**)&(d_coloredNodes), n * sizeof(bool)));
	hipMemcpy(d_coloring, coloring, n * sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(d_coloredNodes, coloredNodes, n * sizeof(bool), hipMemcpyHostToDevice);

	// Calculate inbound counters
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);
	uint* inboundCounts;
	CHECK(hipMalloc((void**)&inboundCounts, n * sizeof(uint)));
	hipMemset(inboundCounts, 0, n * sizeof(uint));
	calculateInbounds << <gridDim, blockDim >> > (d_graphStruct, inboundCounts, d_priorities, n);
	hipDeviceSynchronize();

	// inizialize bitmaps, every node has a bitmap with a length of inbound edges + 1
	uint bitCount = (n + (int)(edgeCount + 1) / 2);
	uint* bitmapIndex = (uint*)malloc((n + 1) * sizeof(uint));
	uint* h_InboundCounts;
	h_InboundCounts = (uint*)malloc(n * sizeof(uint));
	hipMemcpy(h_InboundCounts, inboundCounts, n * sizeof(uint), hipMemcpyDeviceToHost);
	bitmapIndex[0] = 0;
	for (int i = 1; i < n + 1; i++) // can be done in parallel with scan algorithm
		bitmapIndex[i] = bitmapIndex[i - 1] + h_InboundCounts[i - 1] + 1;
	bool* d_bitmaps;
	uint* d_bitmapIndex;
	CHECK(hipMalloc((void**)&d_bitmaps, bitCount * sizeof(bool)));
	CHECK(hipMalloc((void**)&d_bitmapIndex, (n + 1) * sizeof(uint)));
	hipMemset(d_bitmaps, 1, bitCount * sizeof(bool));
	hipMemcpy(d_bitmapIndex, bitmapIndex, (n + 1) * sizeof(uint), hipMemcpyHostToDevice);
	delete(bitmapIndex);
	delete(h_InboundCounts);

	// Alloc buffer needed to synchronize the coloring
	unsigned* buffer;
	hipMalloc((void**)&buffer, n * sizeof(unsigned));
	hipMemset(buffer, 0, n * sizeof(unsigned));
	bool* filledBuffer;
	hipMalloc((void**)&filledBuffer, n * sizeof(bool));
	hipMemset(filledBuffer, 0, n * sizeof(bool));

	int iterationCount = 0;
	bool* uncoloredFlag = (bool*)malloc(sizeof(bool));
	*uncoloredFlag = true;
	bool* d_uncoloredFlag;
	hipMalloc((void**)&d_uncoloredFlag, sizeof(bool));
	double lap = getLapTime();
	std::cout << "Initialization: " << lap << std::endl;
	while (*uncoloredFlag) {
		*uncoloredFlag = false;
		hipMemcpy(d_uncoloredFlag, uncoloredFlag, sizeof(bool), hipMemcpyHostToDevice);
		colorWithInboundCountersBitmaps << <gridDim, blockDim >> > (d_coloring, d_coloredNodes, d_graphStruct, inboundCounts, buffer, filledBuffer, d_bitmaps, d_bitmapIndex, d_uncoloredFlag);
		hipDeviceSynchronize();
		applyBufferWithInboundCountersBitmaps << <gridDim, blockDim >> > (d_coloring, d_coloredNodes, d_graphStruct, d_priorities, inboundCounts, buffer, filledBuffer, d_bitmaps, d_bitmapIndex);
		hipDeviceSynchronize();
		hipMemcpy(uncoloredFlag, d_uncoloredFlag, sizeof(bool), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		iterationCount++;
	}
	lap = getLapTime();
	std::cout << "Processing: " << lap << std::endl;

	//hipMemcpy(coloring, d_coloring, sizeof(Coloring), hipMemcpyDeviceToHost);
	hipMemcpy(coloring, d_coloring, n * sizeof(uint), hipMemcpyDeviceToHost);
	hipMemcpy(coloredNodes, d_coloredNodes, n * sizeof(bool), hipMemcpyDeviceToHost);

	Coloring* coloringStruct = (Coloring*)malloc(sizeof(Coloring));
	coloringStruct->coloring = coloring;
	coloringStruct->coloredNodes = coloredNodes;
	coloringStruct->iterationCount = iterationCount;

	// Free
	hipFree(inboundCounts);
	hipFree(buffer);
	hipFree(filledBuffer);
	hipFree(d_coloring);
	hipFree(d_coloredNodes);
	hipFree(d_bitmaps);
	hipFree(d_bitmapIndex);
	return coloringStruct;
}
