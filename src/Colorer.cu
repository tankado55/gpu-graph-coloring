#include "hip/hip_runtime.h"
﻿
#include ""
#include <iostream>
#include "graph/graph_d.h"
#include "utils/common.h"
#include <hip/hip_cooperative_groups.h>

#include "colorer.h"

#define THREADxBLOCK 128

Colorer::Colorer(Graph* graph)
{
	m_Graph = graph;
	m_GraphStruct = graph->getStruct();
	CHECK(hipMallocManaged(&m_Coloring, sizeof(Coloring)));
	m_Coloring->uncoloredFlag = true;
	m_Coloring->numOfColors = 0;

	uint n = m_GraphStruct->nodeCount;

	CHECK(hipMallocManaged(&m_Coloring->coloring, n * sizeof(uint)));
	memset(m_Coloring->coloring, 0, n * sizeof(uint));
	CHECK(hipMallocManaged(&m_Coloring->coloredNodes, n * sizeof(bool)));
	memset(m_Coloring->coloredNodes, 0, n * sizeof(bool));

	//init inbound counts
	CHECK(hipMallocManaged(&m_InboundCounts, n * sizeof(uint)));
}

Colorer::~Colorer(){
	hipFree(m_InboundCounts);
}

__global__ void initLDF(GraphStruct* graphStruct, int* inboundCounts, int n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;

	uint degree = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];
	printf("node(%d [myDegree: %d] \n", idx, degree);

	inboundCounts[idx] = 0;
	for (uint i = 0; i < degree; ++i)
	{
		uint neighID = graphStruct->neighs[graphStruct->neighIndex[idx] + i];
		uint neighDegree = graphStruct->neighIndex[neighID + 1] - graphStruct->neighIndex[neighID]; // ottimizzabile su CPU
		if (degree > neighDegree)
		{
			atomicAdd(&inboundCounts[neighID], 1);
			printf(" atomicAdd node(%d -> %d [count: %d] \n", idx, neighID, inboundCounts[neighID]);
		}
		else if (degree == neighDegree && idx > neighID)
		{
			atomicAdd(&inboundCounts[neighID], 1);
			printf(" atomicAdd node(%d -> %d [count: %d] \n", idx, neighID, inboundCounts[neighID]);
		}
	}
}

__global__ void initLDF2(GraphStruct* graphStruct, uint* inboundCounts, int n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;

	uint degree = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];
	//printf("node(%d [myDegree: %d] \n", idx, degree);

	inboundCounts[idx] = 0;
	for (uint i = 0; i < degree; ++i) //TODO: ciclo inutile, basta mettere piú 1 a ogni elemento della lista
	{
		uint neighID = graphStruct->neighs[graphStruct->neighIndex[idx] + i];
		
		atomicAdd(&inboundCounts[neighID], 1);
		//printf(" atomicAdd node(%d -> %d [count: %d] \n", idx, neighID, inboundCounts[neighID]);
		
	}
}

__global__ void findISLDF(Coloring* coloring, GraphStruct* graphStruct, bool* bitmaps, uint* bitmapIndex, uint* inboundCounts)
{
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= graphStruct->nodeCount) //é giusto
		return;

	if (coloring->coloredNodes[idx])
		return;

	//printf("GPU - I'm %d, myInbound: %d\n", idx, inboundCounts[idx]);

	uint offset = graphStruct->neighIndex[idx];
	uint deg = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];

	if (inboundCounts[idx] == 0) // Ready node
	{
		int colorCount = bitmapIndex[idx + 1] - bitmapIndex[idx];
		printf("I'm %d, total colors: %d\n", idx, colorCount);

		int bestColor = colorCount;
		for (int i = 0; i < colorCount; ++i)
		{
			if (bitmaps[bitmapIndex[idx] + i])
			{
					bestColor = i;
					break;
			}
		}
		coloring->coloring[idx] = bestColor;
		coloring->coloredNodes[idx] = true;
		printf("colored: %d, best color: %d: \n", idx, coloring->coloring[idx]);
		
		for (uint i = 0; i < deg; i++) {
			uint neighID = graphStruct->neighs[offset + i];
			if (!coloring->coloredNodes[neighID])
			{
				atomicAdd(&inboundCounts[neighID], -1);
				bitmaps[bitmapIndex[neighID] + bestColor] = 0;
			}
		}
	}
	else
	{
		coloring->uncoloredFlag = true;
	}
}

Coloring* Colorer::LDFColoring()
{
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((m_GraphStruct->nodeCount + blockDim.x - 1) / blockDim.x, 1, 1);
	
	// Init DAG TODO: refactorare
	GraphStruct* dag;
	CHECK(hipMallocManaged(&dag, sizeof(GraphStruct)));
	CHECK(hipMallocManaged(&(dag->neighIndex), (m_GraphStruct->nodeCount + 1) * sizeof(int)));
	CHECK(hipMallocManaged(&(dag->neighs), (m_GraphStruct->edgeCount+1)/2 * sizeof(int)));
	m_Graph->getLDFDag(dag);

	//initLDF <<<gridDim, blockDim>>> (m_GraphStruct, m_InboundCounts, m_GraphStruct->nodeCount);
	initLDF2 <<<gridDim, blockDim>>> (dag, m_InboundCounts, m_GraphStruct->nodeCount);
	hipDeviceSynchronize();

	// inizialize bitmaps
	// Every node has a bitmap with a length of inbound edges + 1
	bool* bitmaps;
	uint bitCount = (m_GraphStruct->nodeCount + (int)(m_GraphStruct->edgeCount + 1) / 2);
	CHECK(hipMallocManaged(&(bitmaps), bitCount * sizeof(bool)));
	memset(bitmaps, 1, bitCount * sizeof(bool));

	uint* bitmapIndex;
	CHECK(hipMallocManaged(&bitmapIndex, (m_GraphStruct->nodeCount + 1) * sizeof(uint)));
	hipDeviceSynchronize();
	bitmapIndex[0] = 0;
	for (int i = 1; i < m_GraphStruct->nodeCount + 1; i++)
		bitmapIndex[i] = bitmapIndex[i - 1] + m_InboundCounts[i - 1] + 1; //this info should be taken by the dag and the inbound should be only in gpu mem

	uint iterationCount = 0;
	while (m_Coloring->uncoloredFlag) {
		m_Coloring->uncoloredFlag = false;
		iterationCount++;
		printf("------------ Sequential iteration: %d \n", iterationCount);
		int deb_inBoundSum = 0;
		for (int i = 0; i < m_GraphStruct->nodeCount; ++i)
		{
			deb_inBoundSum += m_InboundCounts[i];
		}
		printf("------------ inboundsum: %d \n", deb_inBoundSum);
		printf("edges: %d", m_GraphStruct->edgeCount);
		int deb_ready = 0;
		for (int i = 0; i < m_GraphStruct->nodeCount; ++i)
		{
			if (m_InboundCounts[i] == 0 && m_Coloring->coloredNodes[i] == false)
				++deb_ready;
		}
		if (deb_ready == 0)
			printf("------------ ready: %d \n", deb_ready);
		findISLDF <<< gridDim, blockDim >>> (m_Coloring, dag, bitmaps, bitmapIndex, m_InboundCounts);
		hipDeviceSynchronize();
	}

	return m_Coloring;
}

Coloring* Colorer::RandomPriorityColoringCPUSequential()
{
	// DAG
	Graph dag(Graph::MemoryEnum::HostAllocated);
	m_Graph->BuildRandomDAG(dag);

	// temp data inizialization
	uint bitCount = (m_GraphStruct->nodeCount + (int)(m_GraphStruct->edgeCount + 1) / 2);
	std::vector<bool> bitmaps(bitCount, true);
	std::vector<uint> bitmapIndex(m_GraphStruct->nodeCount + 1);
	std::vector<uint> inboundCounts(m_GraphStruct->nodeCount, 0);
	GraphStruct* dagStruct = dag.getStruct();
	for (int i = 0; i < dag.GetEdgeCount(); ++i)
		inboundCounts[dagStruct->neighs[i]]++;
	for (int i = 1; i < m_GraphStruct->nodeCount + 1; i++)
		bitmapIndex[i] = bitmapIndex[i - 1] + m_InboundCounts[i - 1] + 1;

	// JP Coloring
	m_Coloring->numOfColors = 0;
	while (m_Coloring->uncoloredFlag)
	{
		m_Coloring->uncoloredFlag = false;
		for (int i = 0; i < m_GraphStruct->nodeCount; ++i)
		{
			if (m_Coloring->coloring[i])
				continue;

			uint offset = dagStruct->neighIndex[i];
			uint deg = dagStruct->neighIndex[i + 1] - dagStruct->neighIndex[i];

			if (inboundCounts[i] == 0) // Ready node
			{
				int colorCount = bitmapIndex[i + 1] - bitmapIndex[i];
				printf("I'm %d, total colors: %d\n", i, colorCount);

				int bestColor = colorCount;
				for (int j = 0; j < colorCount; ++j)
				{
					if (bitmaps[bitmapIndex[i] + j])
					{
						if (j < bestColor)
						{
							//TODO: find another way
							bestColor = j;
							break;
						}
					}
				}
				m_Coloring->coloring[i] = bestColor;
				m_Coloring->coloredNodes[i] = true;
				printf("colored: %d, best color: %d: \n", i, m_Coloring->coloring[i]);
				if (bestColor > m_Coloring->numOfColors)
				{
					m_Coloring->numOfColors = bestColor; // possibile race, potrei computarlo nella print
				}
				for (uint j = 0; j < deg; j++) {
					uint neighID = dagStruct->neighs[offset + j];
					inboundCounts[neighID]--;
					bitmaps[bitmapIndex[neighID] + bestColor] = 0;
					
				}
			}
			else
			{
				m_Coloring->uncoloredFlag = true;
			}
		}
	}
	return m_Coloring;
}

Coloring* RandomPriorityColoringV2(Graph& graph)
{
	// Alloc and Init returning struct
	Coloring* coloring;
	int n = graph.getStruct()->nodeCount;
	mallocOnHost(coloring, n);
	coloring->uncoloredFlag = true;
	coloring->numOfColors = 0;

	// Parallel DAG
	GraphStruct* dag;
	graph.AllocDagOnDevice(dag);
	// Init DAG parallel





	// temp data inizialization

	return nullptr;
}

void mallocOnHost(Coloring* coloring, unsigned n)
{
	coloring = (Coloring*)malloc(sizeof(Coloring));
	coloring->coloring = (uint*)calloc(n, sizeof(uint));
	coloring->coloredNodes = (bool*)calloc(n, sizeof(bool));
}

Coloring* RandomPriorityColoring(GraphStruct* graphStruct)
{
	// set coloring struct
	Coloring* col;
	CHECK(hipMallocManaged(&col, sizeof(Coloring)));
	uint n = graphStruct->nodeCount;
	col->uncoloredFlag = true;

	// hipMalloc for arrays of struct Coloring
	CHECK(hipMallocManaged(&(col->coloring), n * sizeof(uint)));
	memset(col->coloring, 0, n);

	// allocate space on the GPU for the random states
	hiprandState_t* states;
	uint* weigths;
	hipMalloc((void**)&states, n * sizeof(hiprandState_t));
	hipMalloc((void**)&weigths, n * sizeof(uint));
	dim3 threads(THREADxBLOCK);
	dim3 blocks((graphStruct->nodeCount + threads.x - 1) / threads.x, 1, 1);
	uint seed = 0;
	init <<< blocks, threads >>> (seed, states, weigths, n);
	hipDeviceSynchronize();
	// start coloring (dyn. parall.)
	LubyJPcolorer(col, graphStruct, weigths);

	hipFree(states);
	hipFree(weigths);
	return col;
}

/**
 * find an IS
 */
__global__ void findIS(Coloring* col, GraphStruct* graphStruct, uint* weights) {
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= graphStruct->nodeCount)
		return;

	if (col->coloring[idx])
		return;

	uint offset = graphStruct->neighIndex[idx];
	uint deg = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];

	bool candidate = true;
	for (uint j = 0; j < deg; j++) {
		uint neighID = graphStruct->neighs[offset + j];
		if (!col->coloring[neighID] &&
			((weights[idx] < weights[neighID]) ||
				((weights[idx] == weights[neighID]) && idx < neighID))) {
			candidate = false;
		}
	}
	if (candidate) {
		col->coloring[idx] = col->numOfColors;
	}
	else
		col->uncoloredFlag = true;
}

/**
 *  this GPU kernel takes an array of states, and an array of ints, and puts a random int into each
 */
__global__ void init(uint seed, hiprandState_t* states, uint* numbers, uint n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > n)
		return;
	hiprand_init(seed, idx, 0, &states[idx]);
	numbers[idx] = hiprand(&states[idx]) % n * n;
}






/**
 * Luby IS & Jones−Plassmann colorer
 */
void LubyJPcolorer(Coloring* col, GraphStruct* graphStruct, uint* weights) {
	dim3 threads(THREADxBLOCK);
	dim3 blocks((graphStruct->nodeCount + threads.x - 1) / threads.x, 1, 1);

	// loop on ISs covering the graph
	col->numOfColors = 0;
	while (col->uncoloredFlag) {
		col->uncoloredFlag = false;
		col->numOfColors++;
		findIS <<< blocks, threads >>> (col, graphStruct, weights);
		hipDeviceSynchronize();
	}
}


/**
 * Print the graph (verbose = 1 for "verbose print")
 * @param verbose print the complete graph
 */
void printColoring(Coloring* col, GraphStruct* graphStruct, bool verbose) {
	unsigned n = graphStruct->nodeCount;
	std::cout << "** Graph (num node: " << n << ", num edges: " << graphStruct->edgeCount << ")" << std::endl;
	std::cout << "** Coloring (num colors: " << col->numOfColors + 1 << ")" << std::endl;
	if (verbose) {
		for (uint i = 0; i <= col->numOfColors; i++) {
			std::cout << "   color(" << i << ")" << "-> ";
			for (uint j = 0; j < n; j++)
				if (col->coloring[j] == i)
					std::cout << j << " ";
			std::cout << "\n";
		}
		std::cout << "\n";
	}
}

