#include "hip/hip_runtime.h"
#include <iostream>
#include "IncidenceColorer.h"
#include ""
#include <hip/hip_cooperative_groups.h>

__global__ void applyBufferIncidence(uint* coloring, bool* isColored, GraphStruct* graphStruct, uint* buffer, 
	bool* filledBuffer, uint* priorities, bool* bitmaps, uint* bitmapIndex, unsigned n)
{
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= n)
		return;

	if (isColored[idx])
		return;

	if (!filledBuffer[idx])
		return;

	coloring[idx] = buffer[idx];
	isColored[idx] = true;
	filledBuffer[idx] = false;

	uint offset = graphStruct->neighIndex[idx];
	uint deg = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];

	for (uint i = 0; i < deg; i++)
	{
		uint neighID = graphStruct->neighs[offset + i];

		atomicAdd(&priorities[neighID], 1);
		int neighColorCount = bitmapIndex[neighID + 1] - bitmapIndex[neighID];
		if (buffer[idx] < neighColorCount)
			bitmaps[bitmapIndex[neighID] + buffer[idx]] = 0;
	}
}

Coloring* IncidenceColorer::color(Graph& graph)
{
	setStartTime();

	// Init
	unsigned n = graph.GetNodeCount();
	int edgeCount = graph.GetEdgeCount();
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);
	GraphStruct* d_graphStruct;
	graph.getDeviceStruct(d_graphStruct);

	// Alloc and Init returning struct
	double start = seconds();
	uint* coloring = (uint*)malloc(n * sizeof(uint));
	bool* isColored = (bool*)malloc(n * sizeof(bool));
	memset(coloring, 0, n * sizeof(uint));
	memset(isColored, 0, n * sizeof(bool));
	uint* d_coloring;
	bool* d_isColored;
	CHECK(hipMalloc((void**)&(d_coloring), n * sizeof(uint)));
	CHECK(hipMalloc((void**)&(d_isColored), n * sizeof(bool)));
	hipMemcpy(d_coloring, coloring, n * sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(d_isColored, isColored, n * sizeof(bool), hipMemcpyHostToDevice);

	// Incidence priority
	uint* d_priorities;
	hipMalloc((void**)&d_priorities, n * sizeof(uint));
	hipMemset(d_priorities, 0, n * sizeof(uint));

	// inizialize bitmaps
	bool* bitmaps;
	uint bitCount = (n + edgeCount);
	CHECK(hipMallocManaged(&(bitmaps), bitCount * sizeof(bool)));
	memset(bitmaps, 1, bitCount * sizeof(bool));
	uint* bitmapIndex;
	CHECK(hipMallocManaged(&bitmapIndex, (n + 1) * sizeof(uint)));
	bitmapIndex[0] = 0;
	GraphStruct* graphStruct = graph.getStruct();
	for (int i = 1; i < n + 1; i++) // can be paralelized with a scan
	{
		int prevDeg = graphStruct->neighIndex[i] - graphStruct->neighIndex[i - 1];
		bitmapIndex[i] = bitmapIndex[i - 1] + prevDeg + 1;
	}

	// Alloc buffer needed to synchronize the coloring
	uint* buffer;
	hipMalloc((void**)&buffer, n * sizeof(uint));
	hipMemset(buffer, 0, n * sizeof(uint));
	bool* filledBuffer;
	hipMalloc((void**)&filledBuffer, n * sizeof(bool));
	hipMemset(filledBuffer, 0, n * sizeof(bool));

	// Main algo
	int iterationCount = 0;
	bool* uncoloredFlag = (bool*)malloc(sizeof(bool));
	*uncoloredFlag = true;
	bool* d_uncoloredFlag;
	hipMalloc((void**)&d_uncoloredFlag, sizeof(bool));
	double lap = getLapTime();
	std::cout << "Initialization: " << lap << std::endl;
	start = seconds();
	while (*uncoloredFlag) {
		*uncoloredFlag = false;
		hipMemcpy(d_uncoloredFlag, uncoloredFlag, sizeof(bool), hipMemcpyHostToDevice);
		colorWithoutInbounds <<<gridDim, blockDim >>> (d_isColored, d_graphStruct, buffer, filledBuffer, bitmaps, bitmapIndex, d_priorities, d_uncoloredFlag);
		hipDeviceSynchronize();
		hipMemcpy(uncoloredFlag, d_uncoloredFlag, sizeof(bool), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		applyBufferIncidence <<<gridDim, blockDim >>> (
			d_coloring, d_isColored, d_graphStruct, buffer, filledBuffer, d_priorities, bitmaps, bitmapIndex, n);
		hipDeviceSynchronize();
		hipMemcpy(uncoloredFlag, d_uncoloredFlag, sizeof(bool), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		iterationCount++;
	}
	lap = getLapTime();
	std::cout << "Processing: " << lap << std::endl;

	//copy and build results
	hipMemcpy(coloring, d_coloring, n * sizeof(uint), hipMemcpyDeviceToHost);
	hipMemcpy(isColored, d_isColored, n * sizeof(bool), hipMemcpyDeviceToHost);
	Coloring* coloringStruct = (Coloring*)malloc(sizeof(Coloring));
	coloringStruct->coloring = coloring;
	coloringStruct->coloredNodes = isColored;
	coloringStruct->iterationCount = iterationCount;

	// Free
	hipFree(buffer);
	hipFree(filledBuffer);
	hipFree(d_coloring);

	return coloringStruct;
}
