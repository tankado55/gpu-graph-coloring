#include "hip/hip_runtime.h"
#include <iostream>
#include "RandomPriorityColorer.h"

__global__ void InitRandomPriorities(uint seed, hiprandState_t* states, uint* priorities, uint n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n)
		return;
	hiprand_init(seed, idx, 0, &states[idx]);
	priorities[idx] = hiprand(&states[idx]) % n * n;
}

uint* RandomPriorityColorer::calculatePriority(Graph& graph, GraphStruct* d_graphStruct)
{
	int n = graph.GetNodeCount();
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);

	uint* d_priorities;
	hipMalloc((void**)&d_priorities, graph.GetNodeCount() * sizeof(uint));

	hiprandState_t* states;
	hipMalloc((void**)&states, n * sizeof(hiprandState_t));
	uint seed = 0;

	InitRandomPriorities <<<gridDim, blockDim >>> (seed, states, d_priorities, n);

	return d_priorities;
}

Coloring* RandomPriorityColorer::color(Graph& graph)
{
	// Init
	int n = graph.GetNodeCount();
	int edgeCount = graph.GetEdgeCount();

	std::cout << "Copying graph to device ..." << std::endl;
	GraphStruct* d_graphStruct;
	graph.copyToDevice(d_graphStruct);

	// Alloc and Init returning struct
	double start = seconds();
	uint* coloring = (uint*)malloc(n * sizeof(uint));
	bool* coloredNodes = (bool*)malloc(n * sizeof(bool));
	memset(coloring, 0, n * sizeof(uint));
	memset(coloredNodes, 0, n * sizeof(bool));
	uint* d_coloring;
	bool* d_coloredNodes;
	CHECK(hipMalloc((void**)&(d_coloring), n * sizeof(uint)));
	CHECK(hipMalloc((void**)&(d_coloredNodes), n * sizeof(bool)));
	hipMemcpy(d_coloring, coloring, n * sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(d_coloredNodes, coloredNodes, n * sizeof(bool), hipMemcpyHostToDevice);

	// Generate priorities using degrees
	uint* d_priorities = calculatePriority(graph, d_graphStruct);

	// Calculate inbound counters
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);
	uint* inboundCounts;
	CHECK(hipMalloc((void**)&inboundCounts, n * sizeof(uint)));
	hipMemset(inboundCounts, 0, n * sizeof(uint));
	calculateInbounds << <gridDim, blockDim >> > (d_graphStruct, inboundCounts, d_priorities, n);
	hipDeviceSynchronize();

	// inizialize bitmaps, every node has a bitmap with a length of inbound edges + 1 TODO: alloc on gpu
	// vision: allocare tutto in un array come al solito ma serve la prefix sum
	// alternativa1: sequenziale O(n)
	uint bitCount = (n + (int)(edgeCount + 1) / 2);
	uint* bitmapIndex = (uint*)malloc((n + 1) * sizeof(uint));
	uint* h_InboundCounts;
	h_InboundCounts = (uint*)malloc(n * sizeof(uint));
	hipMemcpy(h_InboundCounts, inboundCounts, n * sizeof(uint), hipMemcpyDeviceToHost);
	bitmapIndex[0] = 0;
	for (int i = 1; i < n + 1; i++)
		bitmapIndex[i] = bitmapIndex[i - 1] + h_InboundCounts[i - 1] + 1; //the inbound should be only in gpu mem TODO: parallelize with scan
	bool* d_bitmaps;
	uint* d_bitmapIndex;
	CHECK(hipMalloc((void**)&d_bitmaps, bitCount * sizeof(bool)));
	CHECK(hipMalloc((void**)&d_bitmapIndex, (n + 1) * sizeof(uint)));
	hipMemset(d_bitmaps, 1, bitCount * sizeof(bool));
	hipMemcpy(d_bitmapIndex, bitmapIndex, (n + 1) * sizeof(uint), hipMemcpyHostToDevice);
	delete(bitmapIndex);

	// Alloc buffer needed to synchronize the coloring
	unsigned* buffer;
	hipMalloc((void**)&buffer, n * sizeof(unsigned));
	hipMemset(buffer, 0, n * sizeof(unsigned));
	bool* filledBuffer;
	hipMalloc((void**)&filledBuffer, n * sizeof(bool));
	hipMemset(filledBuffer, 0, n * sizeof(bool));

	// Color TODO: tieni il flag sulla gpu e itera con gli stream
	int iterationCount = 0;
	bool* uncoloredFlag = (bool*)malloc(sizeof(bool));
	*uncoloredFlag = true;
	bool* d_uncoloredFlag;
	hipMalloc((void**)&d_uncoloredFlag, sizeof(bool));
	double stop = seconds();
	std::cout << "Initialization: " << elapsedTime(start, stop) << std::endl;
	start = seconds();
	while (*uncoloredFlag) {
		*uncoloredFlag = false;
		hipMemcpy(d_uncoloredFlag, uncoloredFlag, sizeof(bool), hipMemcpyHostToDevice);
		colorWithInboundCountersBitmaps << <gridDim, blockDim >> > (d_coloring, d_coloredNodes, d_graphStruct, inboundCounts, buffer, filledBuffer, d_bitmaps, d_bitmapIndex, d_uncoloredFlag);
		hipDeviceSynchronize();
		applyBufferWithInboundCountersBitmaps << <gridDim, blockDim >> > (d_coloring, d_coloredNodes, d_graphStruct, d_priorities, inboundCounts, buffer, filledBuffer, d_bitmaps, d_bitmapIndex);
		hipDeviceSynchronize();
		hipMemcpy(uncoloredFlag, d_uncoloredFlag, sizeof(bool), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		iterationCount++;
	}
	stop = seconds();
	std::cout << "Processing: " << elapsedTime(start, stop) << std::endl;


	// Free
	hipFree(d_priorities);
	hipFree(inboundCounts);
	hipFree(buffer);
	hipFree(filledBuffer);

	//hipMemcpy(coloring, d_coloring, sizeof(Coloring), hipMemcpyDeviceToHost);
	hipMemcpy(coloring, d_coloring, n * sizeof(uint), hipMemcpyDeviceToHost);
	hipMemcpy(coloredNodes, d_coloredNodes, n * sizeof(bool), hipMemcpyDeviceToHost);

	Coloring* coloringStruct = (Coloring*)malloc(sizeof(Coloring));
	coloringStruct->coloring = coloring;
	coloringStruct->coloredNodes = coloredNodes;
	coloringStruct->iterationCount = iterationCount;
	return coloringStruct;
}
