#include "hip/hip_runtime.h"
#include <iostream>
#include "SmallestDegreeLast.h"
#include "utils/common.h"
#include "utils/MyDebug.h"

__global__ void assignPriority(uint* priorities, GraphStruct* graphStruct, double avgDeg, uint priority, int* remainingCount, int* sumDeg)
{
    uint idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= graphStruct->nodeCount)
		return;
    if (priorities[idx])
        return;

    uint deg = graphStruct->neighIndex[idx + 1] - graphStruct->neighIndex[idx];

	uint offset = graphStruct->neighIndex[idx];
	uint currentDeg = 0;
	for (uint i = 0; i < deg; ++i)
	{
		int neighId = graphStruct->neighs[offset + i];
		if (priorities[neighId] == 0 || priorities[neighId] == priority)
		{
			currentDeg++;
		}
	}

    if (currentDeg <= avgDeg)
    {
        priorities[idx] = priority;
        atomicSub(remainingCount, 1);
        //atomicSub(sumDeg, deg); // not properly correct, possible solution: keep a buffer of current deg and sum it in parallel after each step
    }
}

uint* SmallestDegreeLast::calculatePriority(Graph& graph, GraphStruct* d_graphStruct)
{
    int n = graph.GetNodeCount();
    dim3 blockDim(THREADxBLOCK);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);

    double avgDeg = 0;
    
    int* sumDeg;
    CHECK(hipMallocManaged(&(sumDeg), sizeof(int)));
    *sumDeg = graph.GetEdgeCount();

    int* remainingCount;
    CHECK(hipMallocManaged(&(remainingCount), sizeof(int)));
    *remainingCount = graph.GetNodeCount();

    uint* d_priorities;
    hipMalloc((void**)&d_priorities, graph.GetNodeCount() * sizeof(uint));

    int i = 1;
    while (*remainingCount > 0) // TODO: if I don't use the average I can use a flag
    {
        //avgDeg = *sumDeg / *remainingCount;
		avgDeg++;
		while (true)
		{
			int prevRemainingCount = *remainingCount;
			assignPriority << <gridDim, blockDim >> > (d_priorities, d_graphStruct, avgDeg, i, remainingCount, sumDeg);
			hipDeviceSynchronize();
			if (prevRemainingCount == *remainingCount)
				break;
			++i;
		}
    }
    return d_priorities;
}

Coloring* SmallestDegreeLast::color(Graph& graph)
{
	// Init
	int n = graph.GetNodeCount();
	int edgeCount = graph.GetEdgeCount();

	std::cout << "Copying graph to device ..." << std::endl;
	GraphStruct* d_graphStruct;
	graph.copyToDevice(d_graphStruct);

	// Alloc and Init returning struct
	double start = seconds();
	uint* coloring = (uint*)malloc(n * sizeof(uint));
	bool* coloredNodes = (bool*)malloc(n * sizeof(bool));
	memset(coloring, 0, n * sizeof(uint));
	memset(coloredNodes, 0, n * sizeof(bool));
	uint* d_coloring;
	bool* d_coloredNodes;
	CHECK(hipMalloc((void**)&(d_coloring), n * sizeof(uint)));
	CHECK(hipMalloc((void**)&(d_coloredNodes), n * sizeof(bool)));
	hipMemcpy(d_coloring, coloring, n * sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(d_coloredNodes, coloredNodes, n * sizeof(bool), hipMemcpyHostToDevice);

	// Generate priorities using degrees
	uint* d_priorities = calculatePriority(graph, d_graphStruct);

	// Calculate inbound counters
	dim3 blockDim(THREADxBLOCK);
	dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1, 1);
	uint* inboundCounts;
	uint* outboundCounts;
	CHECK(hipMalloc((void**)&inboundCounts, n * sizeof(uint)));
	CHECK(hipMalloc((void**)&outboundCounts, n * sizeof(uint)));
	hipMemset(inboundCounts, 0, n * sizeof(uint));
	hipMemset(outboundCounts, 0, n * sizeof(uint));
	calculateInbounds << <gridDim, blockDim >> > (d_graphStruct, inboundCounts, d_priorities, n, outboundCounts);
	hipDeviceSynchronize();

	// inizialize bitmaps, every node has a bitmap with a length of inbound edges + 1 TODO: alloc on gpu
	// vision: allocare tutto in un array come al solito ma serve la prefix sum
	// alternativa1: sequenziale O(n)
	bool* bitmaps;
	uint bitCount = (n + (int)(edgeCount + 1) / 2);
	CHECK(hipMallocManaged(&(bitmaps), bitCount * sizeof(bool)));
	memset(bitmaps, 1, bitCount * sizeof(bool));
	uint* bitmapIndex;
	CHECK(hipMallocManaged(&bitmapIndex, (n + 1) * sizeof(uint)));
	uint* h_InboundCounts;
	h_InboundCounts = (uint*)malloc(n * sizeof(uint));
	hipMemcpy(h_InboundCounts, inboundCounts, n * sizeof(uint), hipMemcpyDeviceToHost);
	bitmapIndex[0] = 0;
	for (int i = 1; i < n + 1; i++)
		bitmapIndex[i] = bitmapIndex[i - 1] + h_InboundCounts[i - 1] + 1; //the inbound should be only in gpu mem TODO: parallelize with scan

	// Alloc buffer needed to synchronize the coloring
	unsigned* buffer;
	hipMalloc((void**)&buffer, n * sizeof(unsigned));
	hipMemset(buffer, 0, n * sizeof(unsigned));
	bool* filledBuffer;
	hipMalloc((void**)&filledBuffer, n * sizeof(bool));
	hipMemset(filledBuffer, 0, n * sizeof(bool));

	// DEBUG
	uint* h_priorities = (uint*)malloc(n * sizeof(uint));
	//hipMemcpy(h_priorities, priorities, n * sizeof(uint), hipMemcpyDeviceToHost);

	// Color TODO: tieni il flag sulla gpu e itera con gli stream
	int iterationCount = 0;
	bool* uncoloredFlag = (bool*)malloc(sizeof(bool));
	*uncoloredFlag = true;
	bool* d_uncoloredFlag;
	hipMalloc((void**)&d_uncoloredFlag, sizeof(bool));
	double stop = seconds();
	std::cout << "Initialization: " << elapsedTime(start, stop) << std::endl;
	start = seconds();
	while (*uncoloredFlag) {
		*uncoloredFlag = false;
		hipMemcpy(d_uncoloredFlag, uncoloredFlag, sizeof(bool), hipMemcpyHostToDevice);
		colorWithInboundCountersBitmaps << <gridDim, blockDim >> > (d_coloring, d_coloredNodes, d_graphStruct, inboundCounts, buffer, filledBuffer, bitmaps, bitmapIndex, d_uncoloredFlag);
		hipDeviceSynchronize();
		applyBufferWithInboundCountersBitmaps << <gridDim, blockDim >> > (d_coloring, d_coloredNodes, d_graphStruct, d_priorities, inboundCounts, buffer, filledBuffer, bitmaps, bitmapIndex);
		hipDeviceSynchronize();
		//hipMemcpy(h_priorities, priorities, n * sizeof(uint), hipMemcpyDeviceToHost); //TODO: remove
		hipMemcpy(uncoloredFlag, d_uncoloredFlag, sizeof(bool), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		iterationCount++;
	}
	stop = seconds();
	std::cout << "Processing: " << elapsedTime(start, stop) << std::endl;


	// Free
	hipFree(d_priorities);
	hipFree(inboundCounts);
	hipFree(buffer);
	hipFree(filledBuffer);
	//hipFree(coloring);
	//hipFree(coloring->coloring);
	//hipFree(coloring->coloredNodes);

	//hipMemcpy(coloring, d_coloring, sizeof(Coloring), hipMemcpyDeviceToHost);
	hipMemcpy(coloring, d_coloring, n * sizeof(uint), hipMemcpyDeviceToHost);
	hipMemcpy(coloredNodes, d_coloredNodes, n * sizeof(bool), hipMemcpyDeviceToHost);

	Coloring* coloringStruct = (Coloring*)malloc(sizeof(Coloring));
	coloringStruct->coloring = coloring;
	coloringStruct->coloredNodes = coloredNodes;
	coloringStruct->iterationCount = iterationCount;
	return coloringStruct;
}


